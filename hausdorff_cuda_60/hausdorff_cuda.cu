#include "hip/hip_runtime.h"
/*
* Copyright (C) 2013  Azlos Corporation
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>
*/

/*
* hausdorff.cu
*
*  Created on: Sep 22, 2013
*      Author: Paolo Galbiati
*/

#include "platform_config.h"
#include "adj_path.h"
#include "hausdorff.h"

template <uint32_t BLOCK_SIZE, uint32_t ITERATIONS_PER_THREAD> __global__ void
    hausdorffCUDA(float* res, const path_point_t* p0, const path_point_t* p1, uint32_t points0, uint32_t points1)
{
    const float EARTH_RADIUS = 6371.0f;

    // Block index
    uint32_t bx = blockIdx.x;
    uint32_t by = blockIdx.y;

    // Thread index
    uint32_t tx = threadIdx.x;
    uint32_t ty = threadIdx.y;

    uint32_t  x = bx * BLOCK_SIZE + tx;
    uint32_t  y = ITERATIONS_PER_THREAD * (by * BLOCK_SIZE + ty);

#pragma unroll
    for (uint32_t i = y; i < y + ITERATIONS_PER_THREAD; i++) {
        if ((x < points0) && (i < points1)) {
        	float p0xlat = p0[x].lat;
        	float p0xlon = p0[x].lon;
        	float p1ilat = p1[i].lat;
        	float p1ilon = p1[i].lon;

            float delta_lat = (p1ilat - p0xlat) * 0.5f;
            float delta_lon = (p1ilon - p0xlon) * 0.5f;
            float tmp0 = __sinf(delta_lat);
            float tmp1 = __sinf(delta_lon);

            float a = tmp0 * tmp0 + __cosf(p0xlat) * __cosf(p1ilat) * tmp1 * tmp1;
            float c = 2 * atan2f(sqrtf(a), sqrtf(1 - a));
 
            *(res + x * points1 + i) = EARTH_RADIUS * c;
        }
    }
}

void hausdorffGPU(float* res, const path_point_t* p0, const path_point_t* p1, uint32_t points0, uint32_t points1) {

    const uint32_t BLOCK_SIZE = 32;
    const uint32_t ITERATIONS_PER_THREAD = 1;

    // Setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    uint32_t dim_x = points0 / threads.x;
    uint32_t dim_y = points1 / threads.y;
    if (points0 % threads.x) {
        dim_x += 1;
    }
    if (points1 % threads.y) {
        dim_y += 1;
    }

    if (dim_y % ITERATIONS_PER_THREAD == 0) {
        dim_y = dim_y / ITERATIONS_PER_THREAD;
    }
    else {
        dim_y = dim_y / ITERATIONS_PER_THREAD + 1;
    }

    dim3 grid(dim_x, dim_y);


    hausdorffCUDA<BLOCK_SIZE, ITERATIONS_PER_THREAD><<< grid, threads >>>(res, p0, p1, points0, points1);
}



float hausdorff_gpu::distance_impl(const shared_ptr<gpu_device> gpu,
                        const adj_path& p0,
                        const adj_path& p1) {

	path_point_t* device_data0;
	path_point_t* device_data1;
    uint32_t points0 = p0.get_points_number();
    uint32_t points1 = p1.get_points_number();
    shared_ptr<float> results(new float[points0 * points1]);
    float* results_ptr = results.get();
    float dist = 0.0f;

    /* Allocate GPU buffer */
    if (false
            == gpu->gpu_device_malloc((void**) &device_data0,
            		points0 * (sizeof(path_point_t)))) {
        return dist;
    }
    // Copy path from host memory to GPU buffer.
    if (false
            == gpu->gpu_memcpy(device_data0, p0.get_point(0),
            		points0 * (sizeof(path_point_t)),
                               gpu_memcpy_host_to_device)) {
        return dist;
    }

    /* Allocate GPU buffer */
    if (false
            == gpu->gpu_device_malloc((void**) &device_data1,
            		points1 * (sizeof(path_point_t)))) {
        return dist;
    }
    // Copy path from host memory to GPU buffer.
    if (false
            == gpu->gpu_memcpy(device_data1, p1.get_point(0),
            		points1 * (sizeof(path_point_t)),
                               gpu_memcpy_host_to_device)) {
        return dist;
    }

    /* Allocate GPU buffer */
	uint32_t data_size = (points0 * points1) * sizeof(float);
	float* result_buffer = nullptr;
    
    if (false == gpu->gpu_device_malloc((void**)&result_buffer, data_size)) 
    {
       return dist;
    }

    hausdorffGPU(result_buffer, device_data0, device_data1, points0, points1);

    gpu->gpu_device_synchronize();

    // Copy result from device to host
    if (true == gpu->gpu_memcpy(results.get(), result_buffer, data_size, gpu_memcpy_device_to_host)) 
	{
        dist = maxmin_impl(results.get(), points0, points1);
    }

    gpu->gpu_device_free(device_data0);
    gpu->gpu_device_free(device_data1);
    gpu->gpu_device_free(result_buffer);

    return dist;
}
